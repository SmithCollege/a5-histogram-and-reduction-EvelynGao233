#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void gpuHistogramNonStrided(int *input, int *hist, int n, int bins) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < n) {
        atomicAdd(&hist[input[tid] % bins], 1);
    }
}

int main() {
    int sizes[] = {128, 512, 2048, 4096};
    int bins = 10;

    for (int s = 0; s < 4; s++) {
        int n = sizes[s];
        int *h_input = (int*)malloc(n * sizeof(int));
        int *h_hist = (int*)calloc(bins, sizeof(int));  // Initialize histogram on host

        // Initialize input array on host
        for (int i = 0; i < n; i++) h_input[i] = i % bins;

        int *d_input, *d_hist;
        hipMalloc(&d_input, n * sizeof(int));
        hipMalloc(&d_hist, bins * sizeof(int));

        // Copy input data from host to device
        hipMemcpy(d_input, h_input, n * sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_hist, 0, bins * sizeof(int));  // Initialize histogram on device

        dim3 blockSize(256);
        dim3 gridSize((n + blockSize.x - 1) / blockSize.x);

        // Record start time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Launch kernel for non-strided GPU histogram
        gpuHistogramNonStrided<<<gridSize, blockSize>>>(d_input, d_hist, n, bins);
        hipDeviceSynchronize();

        // Record stop time
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Copy histogram result back to host
        hipMemcpy(h_hist, d_hist, bins * sizeof(int), hipMemcpyDeviceToHost);

        // Print the result and time taken
        printf("Array Size: %d, Time taken (GPU Histogram Non-Strided): %f ms\n", n, milliseconds);
        printf("Histogram:\n");
        for (int i = 0; i < bins; i++) {
            printf("Bin %d: %d\n", i, h_hist[i]);
        }
        printf("\n");

        // Free memory
        free(h_input);
        free(h_hist);
        hipFree(d_input);
        hipFree(d_hist);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    return 0;
}
